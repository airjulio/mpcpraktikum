#include "hip/hip_runtime.h"
/*
 * CMEstimatorGPUSparse.cu
 *
 * Generates a list of indices containing the i, j index of approx. the 
 * k-best confidence measure values. 
 * The List of indices is generated column wise after Cula solved the
 * linear equation system. This class uses the already stored memory
 * of the equation solver and extracts the k best values of the specific
 * column.
 *
 *  Created on: 19.06.2013
 *      Author: Fabian, Armin
 */

#include "GPUSparse.h"
#include "CMEstimatorGPUSparse.h"
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <ctime>
#include <vector>
#include <algorithm> /* std::find */
#include <stdio.h> /* printf */
#include <float.h> /* FLT_MAX */
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <cula_sparse.h>
#include "Tester.h"
#include "Helper.h"

#define CHECK_FOR_CUDA_ERROR 0

#define CUDA_CHECK_ERROR() {							\
    hipError_t err = hipGetLastError();					\
    if (hipSuccess != err) {						\
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
                __FILE__, __LINE__, hipGetErrorString(err) );	\
        exit(EXIT_FAILURE);						\
    }									\
}

inline __int64_t continuousTimeNs()
 {
         timespec now;
         clock_gettime(CLOCK_REALTIME, &now);

         __int64_t result = (__int64_t ) now.tv_sec * 1000000000
                         + (__int64_t ) now.tv_nsec;

         return result;
 }

const int THREADS = 128;

__device__ int d_notWritten;

//Initialize index arrays
static __global__ void initIndexArrays(int* d_idx1, int* d_idx2, int* d_res, int size, unsigned int dim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size)
	{
		d_idx1[idx] = dim+1;
		d_idx2[idx] = dim+1;
		d_res[idx] = 0;
	}
}


//Initialize indices
static __global__ void initKernel(long* gpuIndices, double* x, double* b, const int dim, const int columnIdx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < dim)
	{
		if (columnIdx >= idx || 0 != b[idx]) //diagonal element or known element or upper diagonal matrix element
		{
			gpuIndices[idx] = -1;
			//assign very low value to avoid them getting chosen
			x[idx] = -DBL_MAX;
		}
		else
		{
			//assign index value based on the overall matrix dimension (continuous idx)
			gpuIndices[idx] = columnIdx + idx * dim;
		}
	}
}

//Write indices to respective index arrays
static __global__ void saveIndicesKernel(long* gpuIndices, int* d_idx1, int* d_idx2, int dim, int kBest, int kBestForThisColumn, int currIndexNr)
{
	int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int write_idx = t_idx + currIndexNr;

	if (t_idx < dim && write_idx < kBest && t_idx < kBestForThisColumn)
	{
		int gpuIdx = gpuIndices[t_idx]; //size(gpuIndices) > kBestForThisColumn > t_idx

		if (-1 != gpuIdx)
		{
			int i = gpuIdx/dim;
			int j = gpuIdx%dim;

			d_idx1[write_idx] = i;
			d_idx2[write_idx] = j;
		}
		else
		{
			atomicAdd(&d_notWritten, 1); //save nr of threads not writing anything
		}
	}
}

CMEstimatorGPUSparse::CMEstimatorGPUSparse() {
	lastSize = 0;
	d_idx1 = NULL;
	d_idx2 = NULL;
	d_res = NULL;

	totalTime = 0;

	/* cula initialization */
	//initCula();
	printf("Using random estimator (CMEstimatorGPUSparse)\n");
}


CMEstimatorGPUSparse::~CMEstimatorGPUSparse() {
	//free device pointer
	if (d_idx1 != NULL) hipFree(d_idx1);
	if (d_idx2 != NULL) hipFree(d_idx2);
	if (d_res != NULL) hipFree(d_res);
	printf("Total solver time: %f\n", totalTime*(1/(double)1000000000));
}

int* CMEstimatorGPUSparse::getIdx1Ptr()
{
	return d_idx1;
}

int* CMEstimatorGPUSparse::getIdx2Ptr()
{
	return d_idx2;
}

int* CMEstimatorGPUSparse::getResPtr()
{
	return d_res;
}

void CMEstimatorGPUSparse::initCula()
{
//	//config solver
//	config.relativeTolerance = 1e-6;
//	config.maxIterations = 300;
//	config.maxRuntime = 10;
//
//	culaSparseCreate(&handle); //create library handle
//	culaSparseConfigInit(handle, &config); //initialize values
//	culaSparseCreatePlan(handle, &plan); //create execution plan
//	culaSparseSetCudaDevicePlatform(handle, plan, 0); //use the CUDA-device platform (interprets given pointer as device pointers)
//	culaSparseSetCgSolver(handle, plan, 0); //associate CG solver with the plan
//	culaSparseSetJacobiPreconditioner(handle, plan, 0); //associate jacobi preconditioner with the plan
}

//Allocate device memory for index pointers and clear last used pointers
//(for dynamic change of kBest values index-arays)
void CMEstimatorGPUSparse::initIdxDevicePointers(int size, unsigned int dim)
{
	//clear previous pointers
	if (d_idx1 != NULL) hipFree(d_idx1);
	if (d_idx2 != NULL) hipFree(d_idx2);
	if (d_res != NULL) hipFree(d_res);

	//allocate new device memory
	hipMalloc((void**) &d_idx1, size * sizeof(int));
	hipMalloc((void**) &d_idx2, size * sizeof(int));
	hipMalloc((void**) &d_res, size * sizeof(int));

	//Kernel settings for index array
	int numBlocks = (size + THREADS - 1) / THREADS;
	initIndexArrays<<<numBlocks, THREADS>>>(d_idx1, d_idx2, d_res, size, dim);

//	printf("[ESTIMATOR]: Device index arrays with size %i allocated.\n",size);
}

/*
 * Determines column-wise the best confidence measures of the specific column and saves it indices in two arrays.
 * Returns the number of actually written slots.
 */
int CMEstimatorGPUSparse::determineBestConfMeasures(double* xColumnDevice, double* bColumnDevice, int columnIdx, int dim, int kBest, int kBestForThisColumn, int currIndexNr)
{
	//Allocate index array on GPU
	long* gpuIndices;
	hipMalloc((void**) &gpuIndices, dim * sizeof(long));

	//wrap raw pointer with device pointer
	thrust::device_ptr<long> dp_gpuIndices = thrust::device_pointer_cast(gpuIndices);

	//Kernel settings for index array
	int numBlocks = (dim + THREADS - 1) / THREADS;
	int numThreads = THREADS;

	/* Init indices array such that indices = [-1,1,2,-1,...,dim-1], whereas the respective
	 * diagonal element is -1 as well as elements that are already compared or within the upper
	 * diagonal matrix.
	 * For already known elements (i.e. bColumnDevice[i] != 0), xColumnDevice[i] will be
	 * assigned a very low value to prevent them from getting chosen later.
	 */
	initKernel<<<numBlocks, numThreads>>>(gpuIndices, xColumnDevice, bColumnDevice, dim, columnIdx);
#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//wrap column device pointer
	thrust::device_ptr<double> dp_xColumn = thrust::device_pointer_cast(xColumnDevice);

	//sort x column and index array respectively
	//already known values will be the last ones due to initialization
	thrust::sort_by_key(dp_xColumn, dp_xColumn + dim, dp_gpuIndices, thrust::greater<double>());

//	Helper::printGpuArrayD(xColumnDevice, 10, "Top10");

	//maybe recast pointers? (from thrust)
	//unsigned int * raw_ptr = thrust::raw_pointer_cast(dev_data_ptr);

	//save 'kBestForThisColumn' indices if possible (maybe not enough indices available)
	numBlocks = (kBest + THREADS - 1) / THREADS;
	int notWritten = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(d_notWritten), &notWritten, sizeof(int));
	saveIndicesKernel<<<numBlocks, numThreads>>>(gpuIndices, d_idx1, d_idx2, dim, kBest, kBestForThisColumn, currIndexNr);
	hipMemcpyFromSymbol(&notWritten, HIP_SYMBOL(d_notWritten), sizeof(int));

//	printf("notWritten = %i\n", notWritten);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//free memory
	hipFree(gpuIndices);

	return kBestForThisColumn - notWritten;
}

void CMEstimatorGPUSparse::getKBestConfMeasures(MatrixHandler* T, float* F, int kBest)
{
//	printf("[ESTIMATOR]: Determine kBest confidence measures on GPU (column-wise).\n");

	//invoked only on sparse MatrixHandler
	GPUSparse* T_sparse = dynamic_cast<GPUSparse*> (T);
	unsigned int dim = T_sparse->getDimension();

	//indices cache

	/* if index array size changed since last use, allocate new device memory
	 * with new size and free old device memory. Otherwise reuse device memory.
	 */
	if (kBest != lastSize)
	{
		initIdxDevicePointers(kBest, dim);
		lastSize = kBest;
	}
	int countIndices = 0;

	//set up data for solver
	unsigned int nnz = T_sparse->getNNZ();
	double* d_values = T_sparse->getValueArrayDouble(true);
	int* d_colIdx = T_sparse->getColIdxDevice();
	int* d_rowPtr = T_sparse->getRowPtrDevice();

	//x-vector
	double* d_x;
	hipMalloc((void**) &d_x, dim * sizeof(double));

//	Helper::printGpuArrayD(d_values, nnz, "Values");

	//b-vector
	double* d_b;

	//Reinitialize cula to ensure proper execution
	//config solver
	culaSparseHandle handle;
	culaSparsePlan plan;
	culaSparseConfig config;

	culaSparseCreate(&handle); //create library handle
	culaSparseCreatePlan(handle, &plan); //create execution plan

    culaSparseCudaDeviceOptions platformOpts;
    //use the CUDA-device platform (interprets given pointer as device pointers)
    culaSparseStatus statCula = culaSparseCudaDeviceOptionsInit(handle, &platformOpts);
	platformOpts.deviceId = 0;
	platformOpts.debug = 0;
	statCula = culaSparseSetCudaDevicePlatform(handle, plan, &platformOpts);

	culaSparseConfigInit(handle, &config); //initialize config values
//	config.relativeTolerance = 1e-4;
//	config.maxIterations = 50;
//	config.maxRuntime = 1;
//	config.useBestAnswer = 1;

	culaSparseSetCgSolver(handle, plan, 0); //associate CG solver with the plan
	culaSparseSetJacobiPreconditioner(handle, plan, 0); //associate jacobi preconditioner with the plan

	int noError = 0;
	int solverTrials = 0;
	int determinedIndicesByNow = 0;
	int column = rand() % dim;
	int xBestForThisColumn = kBest*0.001;
	char* colsVisited = (char*)malloc(sizeof(char)*dim);
	memset(colsVisited, 0, dim);
	//printf("[CMESTIMATOR]: Solve Eq. system column by column.\n");
	__int64_t startCula = continuousTimeNs();
	for(int i = 0; i < (dim) && countIndices < kBest; i++) //if enough values are gathered, stop computation
	{
		//0. determine number of best values for this column
		//The bigger i, the less best indices are determined for this column
//		int xBestForThisColumn = ((dim-i)/(0.5*dim*(dim-1))) * kBest;
		if (!xBestForThisColumn) xBestForThisColumn = 1; //at least 1 per column
		//take into account that probably not as many indices as needed can be determined, so try to get them in the next column
		int determineXforThisColumn = xBestForThisColumn + (determinedIndicesByNow - countIndices);
		//1. Compute confidence measure for this column (solve Ax=b)

		d_b = T_sparse->getColumnDouble(column);
		//Helper::printGpuArrayD(d_b, dim, "B");
		colsVisited[column] = 1;

		culaSparseStatus res = computeConfidenceMeasure(handle, plan, config, dim, nnz, d_values, d_rowPtr, d_colIdx, d_x, d_b);

		solverTrials++;

		if(res == culaSparseUnspecifiedError || res == culaSparseRuntimeError || res == culaSparseInteralError)
		{
			//A bad CULA error occurred

			if(res == culaSparseRuntimeError)
			{
				printf("Exiting due to CULA Runtime Error\n");
			}
			else
			{
				printf("Exiting due to CULA internal or unspecified ERROR!\n");
			}

			T_sparse->print();
			exit(EXIT_FAILURE);
		}
		else
		{
			if(res == culaSparseNoError)
				noError++;

//			Helper::printGpuArrayD(d_x, dim, "X");

			//2. get indices of x best confidence measure values
			int writtenIndices = determineBestConfMeasures(d_x, d_b, column, dim, kBest, determineXforThisColumn, countIndices);
			countIndices += writtenIndices;

		}

		//hipFree(d_b);

		determinedIndicesByNow += xBestForThisColumn; // #indices that should have been determined

		column = rand() % dim;
		const int startCol = column;
		while(colsVisited[column] == 1)
		{
			column = (column + 1) % dim;
			if(column == startCol)
				break;
		}

//		printf("Column %i, try to determine %i best values. Actually determined by now %i values\n", i, determineXforThisColumn, countIndices);
	}
	__int64_t solverDiff = continuousTimeNs()-startCula;
	totalTime += solverDiff;
	//printf("Solver time [%i of %i NO ERROR]: %f\n", noError, solverTrials, solverDiff*(1/(double)1000000000));
	printf("%f\t%i\t", solverDiff*(1/(double)1000000000), (solverTrials-noError));
	free(colsVisited);

	//printf("After solving [%i of %i NO ERROR]! Going to sort with thrust\n", noError, solverTrials);
	//sort first index array and second index array respectively
	//wrap device pointers
	thrust::device_ptr<int> dp_idx1 = thrust::device_pointer_cast(d_idx1);
	thrust::device_ptr<int> dp_idx2 = thrust::device_pointer_cast(d_idx2);

	thrust::sort_by_key(dp_idx1, dp_idx1 + kBest, dp_idx2); //sort ascending
#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//clean up the mess
	hipFree(d_x);
	hipFree(d_values);
	culaSparseDestroyPlan(plan);
	culaSparseDestroy(handle);
}

//handles only device pointer.
culaSparseStatus CMEstimatorGPUSparse::computeConfidenceMeasure(culaSparseHandle handle, culaSparsePlan plan, culaSparseConfig config,
															unsigned int dim, unsigned int nnz, double* A, int* rowPtr, int* colIdx, double* x, double* b)
{
	// information returned by the solver
	culaSparseResult result;

	// associate coo data with the plan
	culaSparseSetDcsrData(handle, plan, 0, dim, nnz, A, rowPtr, colIdx, x, b);

	// execute plan
	culaSparseStatus status = culaSparseExecutePlan(handle, plan, &config, &result);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//print if error
	if (culaSparseNoError != status)
	{
		char buffer[512];
		culaSparseGetResultString(handle, &result, buffer, 512);
		printf("%s\n", buffer);
	}



	return status;
}

void CMEstimatorGPUSparse::computeRandomComparisons(MatrixHandler* T, const int k)
{
	printf("0\t0\t");
	GPUSparse* matrix = dynamic_cast<GPUSparse*>(T);
	if (k != lastSize)
	{
		initIdxDevicePointers(k, matrix->getDimension());
		lastSize = k;
	}
	matrix->fillRandomCompareIndices(d_idx1, d_idx2, d_res, k);
}

