#include "hip/hip_runtime.h"
/*
 * CMEstimatorGPUSparse.cu
 *
 * Generates a list of indices containing the i, j index of approx. the 
 * k-best confidence measure values. 
 * The List of indices is generated column wise after Cula solved the
 * linear equation system. This class uses the already stored memory
 * of the equation solver and extracts the k best values of the specific
 * column.
 *
 *  Created on: 19.06.2013
 *      Author: Fabian
 */

#include "GPUSparse.h"
#include "CMEstimatorGPUSparse.h"
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <vector>
#include <algorithm> /* std::find */
#include <stdio.h> /* printf */
#include <float.h> /* FLT_MAX */
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <cula_sparse.h>
#include "Tester.h"

#define CUDA_CHECK_ERROR() {							\
    hipError_t err = hipGetLastError();					\
    if (hipSuccess != err) {						\
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
                __FILE__, __LINE__, hipGetErrorString(err) );	\
        exit(EXIT_FAILURE);						\
    }									\
}

const int THREADS = 64;

__device__ int d_notWritten;

//Initialize index arrays
static __global__ void initIndexArrays(int* d_idx1, int* d_idx2, int* d_res, int size, unsigned int dim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size)
	{
		d_idx1[idx] = dim+1;
		d_idx2[idx] = dim+1;
		d_res[idx] = 0;
	}
}

//Initialize indices
static __global__ void initKernel(long* gpuIndices, float* x, const float* b, const int dim, const int columnIdx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < dim)
	{
		if (columnIdx >= idx || 0 != b[idx]) //diagonal element or known element or upper diagonal matrix element
		{
			gpuIndices[idx] = -1;
			//assign very low value to avoid them getting chosen
			x[idx] = -FLT_MAX;
		}
		else
		{
			//assign index value based on the overall matrix dimension (continuous idx)
			gpuIndices[idx] = columnIdx + idx * dim;
		}
	}
}

//Write indices to respective index arrays
static __global__ void saveIndicesKernel(long* gpuIndices, int* d_idx1, int* d_idx2, int dim, int kBest, int kBestForThisColumn, int currIndexNr)
{
	int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int write_idx = t_idx + currIndexNr;

	if (t_idx < dim && write_idx < kBest)
	{
		int gpuIdx = gpuIndices[t_idx]; //size(gpuIndices) > kBestForThisColumn > t_idx

		if (-1 != gpuIdx)
		{
			int i = gpuIdx/dim;
			int j = gpuIdx%dim;

			d_idx1[write_idx] = i;
			d_idx2[write_idx] = j;
		}
		else
		{
			atomicAdd(&d_notWritten, 1); //save nr of threads not writing anything
		}
	}
}

CMEstimatorGPUSparse::CMEstimatorGPUSparse() {
	lastSize = 0;
	d_idx1 = NULL;
	d_idx2 = NULL;
	d_res = NULL;
}


CMEstimatorGPUSparse::~CMEstimatorGPUSparse() {
	//free device pointer
	if (d_idx1 != NULL) hipFree(d_idx1);
	if (d_idx2 != NULL) hipFree(d_idx2);
	if (d_res != NULL) hipFree(d_res);
}


//Allocate device memory for index pointers and clear last used pointers
//(for dynamic change of kBes values index-arays)
void CMEstimatorGPUSparse::initIdxDevicePointers(int size, unsigned int dim)
{
	//clear previous pointers
	if (d_idx1 != NULL) hipFree(d_idx1);
	if (d_idx2 != NULL) hipFree(d_idx2);
	if (d_res != NULL) hipFree(d_res);

	//allocate new device memory
	hipMalloc((void**) &d_idx1, size * sizeof(int));
	hipMalloc((void**) &d_idx2, size * sizeof(int));
	hipMalloc((void**) &d_res, size * sizeof(int));

	//Kernel settings for index array
	int numBlocks = (size + THREADS - 1) / THREADS;
	dim3 threadBlock(THREADS);
	dim3 blockGrid(numBlocks);
	initIndexArrays<<<blockGrid, threadBlock>>>(d_idx1, d_idx2, d_res, size, dim);

	//todo remove debug printing
	int* testResult1 = new int[size];
	int* testResult2 = new int[size];
	int* testResult3 = new int[size];
	hipMemcpy(testResult1, d_idx1, size*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(testResult2, d_idx2, size*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(testResult3, d_res, size*sizeof(int), hipMemcpyDeviceToHost);
	Tester::printArrayInt(testResult1, size);
	Tester::printArrayInt(testResult2, size);
	Tester::printArrayInt(testResult3, size);

	printf("[ESTIMATOR]: Device index arrays with size %i allocated.\n",size);
}

//todo remove me
Indices* CMEstimatorGPUSparse::getInitializationIndices(MatrixHandler* T, int initNr)
{
	Indices* initIndices = new Indices[initNr];
	std::vector<int> chosenOnes; //max size will be initNr
	int dim = T->getDimension();

	//generate random index
	srand (time(NULL));
	const int MAX_ITERATIONS = dim*(dim/2) + dim; //#elements in upper diagonal matrix + dim

	//generate initialization indices
	for(int i = 0; i < initNr; i++)
	{
		int rIdx = -1;
		int x, y;

		int c = 0;
		do {
			//get random number
			rIdx = rand() % (dim*dim);

			//compute matrix indices with given continuous index sequence
			x = rIdx/dim;
			y = rIdx%dim;
			c++;
		} while ( ((rIdx < 1+(rIdx/dim)+(rIdx/dim)*dim)
					|| (T->getVal(x,y) != 0)
					|| (std::find(chosenOnes.begin(), chosenOnes.end(), rIdx) != chosenOnes.end()))
				&& (c <= MAX_ITERATIONS) );
		/* As long as the random number is not within the upper diagonal matrix w/o diagonal elements
		 * or T(idx) != 0 generate or already in the list of Indices, a new random index but maximal
		 * MAX_ITERAtION times.
		 */

		if (c <= MAX_ITERATIONS) //otherwise initIndices contains -1 per struct definition
		{
			chosenOnes.push_back(rIdx);
			initIndices[i].i = x;
			initIndices[i].j = y;
		}
	}

	return initIndices;
}

//A*x_i = b_i
//todo remove me!
Indices* CMEstimatorGPUSparse::getKBestConfMeasures(float* xColumnDevice, float* bColumnDevice, int columnIdx, int dim, int kBestForThisColumn)
{
	//storage for the kBest indices
	//TODO remove me
	Indices* kBestIndices = new Indices[kBestForThisColumn];

	//Allocate index array on GPU
	long* gpuIndices;
	hipMalloc((void**) &gpuIndices, dim * sizeof(long));
	CUDA_CHECK_ERROR();
	//wrap raw pointer with device pointer
	thrust::device_ptr<long> dp_gpuIndices = thrust::device_pointer_cast(gpuIndices);
	CUDA_CHECK_ERROR();

	//Kernel settings for index array
	int numBlocks = (dim + THREADS - 1) / THREADS;
	dim3 threadBlock(THREADS);
	dim3 blockGrid(numBlocks);

	/* Init indices array such that indices = [-1,1,2,-1,...,dim-1], whereas the respective
	 * diagonal element is -1 as well as elements that are already compared or within the upper
	 * diagonal matrix.
	 * For already known elements (i.e. bColumnDevice[i] != 0), xColumnDevice[i] will be
	 * assigned a very low value to prevent them from getting chosen later.
	 */
	initKernel<<<blockGrid, threadBlock>>>(gpuIndices, xColumnDevice, bColumnDevice, dim, columnIdx);
	CUDA_CHECK_ERROR();

	//wrap column device pointer
	thrust::device_ptr<float> dp_xColumn = thrust::device_pointer_cast(xColumnDevice);
	CUDA_CHECK_ERROR();

	//sort x column and index array respectively
	//already known values will be the last ones due to initialization
	thrust::sort_by_key(dp_xColumn, dp_xColumn + dim, dp_gpuIndices, thrust::greater<float>());
	CUDA_CHECK_ERROR();

	//download device memory
	long* indices = new long[kBestForThisColumn]; //at most kBest indices are needed
	//the first kBest indices are also the best conf. measure values after sorting
	thrust::copy(dp_gpuIndices, dp_gpuIndices + kBestForThisColumn, indices);
	CUDA_CHECK_ERROR();

	//free memory
	hipFree(gpuIndices);

	//build indices list structure
	for(int i = 0; i<kBestForThisColumn; i++)
	{
		long idx = indices[i];
		if (indices[i] > -1)
		{
			kBestIndices[i].i = idx/dim;
			kBestIndices[i].j = idx%dim;
		}
		else
		{
			//after the first index with -1 all following
			//will contain -1.
			break;
		}
		//if some of the indices contained -1, the remaining
		//kBestIndices will contain also -1 as i,j index per
		//struct definition.
	}

	return kBestIndices;
}


/*
 * Determines column-wise the best confidence measures of the specific column and saves it indices in two arrays.
 * Returns the number of actually written slots.
 */
int CMEstimatorGPUSparse::determineBestConfMeasures(float* xColumnDevice, float* bColumnDevice, int columnIdx, int dim, int kBest, int kBestForThisColumn, int currIndexNr)
{
	//Allocate index array on GPU
	long* gpuIndices;
	hipMalloc((void**) &gpuIndices, dim * sizeof(long));
	CUDA_CHECK_ERROR();
	//wrap raw pointer with device pointer
	thrust::device_ptr<long> dp_gpuIndices = thrust::device_pointer_cast(gpuIndices);
	CUDA_CHECK_ERROR();

	//Kernel settings for index array
	int numBlocks = (dim + THREADS - 1) / THREADS;
	dim3 threadBlock(THREADS);
	dim3 blockGrid(numBlocks);

	/* Init indices array such that indices = [-1,1,2,-1,...,dim-1], whereas the respective
	 * diagonal element is -1 as well as elements that are already compared or within the upper
	 * diagonal matrix.
	 * For already known elements (i.e. bColumnDevice[i] != 0), xColumnDevice[i] will be
	 * assigned a very low value to prevent them from getting chosen later.
	 */
	initKernel<<<blockGrid, threadBlock>>>(gpuIndices, xColumnDevice, bColumnDevice, dim, columnIdx);
	CUDA_CHECK_ERROR();

	//wrap column device pointer
	thrust::device_ptr<float> dp_xColumn = thrust::device_pointer_cast(xColumnDevice);
	CUDA_CHECK_ERROR();

	//sort x column and index array respectively
	//already known values will be the last ones due to initialization
	thrust::sort_by_key(dp_xColumn, dp_xColumn + dim, dp_gpuIndices, thrust::greater<float>());
	CUDA_CHECK_ERROR();

	//maybe recast pointers? (from thrust)
	//unsigned int * raw_ptr = thrust::raw_pointer_cast(dev_data_ptr);

	//save 'kBestForThisColumn' indices if possible (maybe not enough indices available)
	numBlocks = (kBest + THREADS - 1) / THREADS;
	dim3 blockGrid2(numBlocks);
	int notWritten = 0;
	hipMemcpyToSymbol(HIP_SYMBOL(d_notWritten), &notWritten, sizeof(int));
	saveIndicesKernel<<<blockGrid2, threadBlock>>>(gpuIndices, d_idx1, d_idx2, dim, kBest, kBestForThisColumn, currIndexNr);
	hipMemcpyFromSymbol(&notWritten, HIP_SYMBOL(d_notWritten), sizeof(int));

	//free memory
	hipFree(gpuIndices);

	return kBestForThisColumn - notWritten;
}



Indices* CMEstimatorGPUSparse::getKBestConfMeasures(MatrixHandler* T, float* F, int kBest)
{
	printf("Determine kBest confidence measures on GPU (column-wise):\n");
	bool newMethod = true;// todo remove me

	//invoked only on sparse matrixhandler
	GPUSparse* T_sparse = dynamic_cast<GPUSparse*> (T);
	unsigned int dim = T_sparse->getDimension();

	//indices cache
	//TODO not needed remove me
	Indices* bestIndices = new Indices[kBest];

	/* if index array size changed since last use, allocate new device memory
	 * with new size and free old device memory. Otherwise reuse device memory.
	 */
	if (kBest != lastSize)
	{
		initIdxDevicePointers(kBest, dim);
		lastSize = kBest;
	}
	int countIndices = 0;

	//set up data for solver
	unsigned int nnz = T_sparse->getNNZ();

	float* d_values = T_sparse->getValueArr(true);

	int* d_colIdx = T_sparse->getColIdxDevice();
	int* d_rowPtr = T_sparse->getRowPtrDevice();

	//GPUSparse::printGpuArrayF(d_values, nnz, "[ESTIMATOR] Values"); CUDA_CHECK_ERROR()
	//GPUSparse::printGpuArray(d_rowPtr, dim+1, "[ESTIMATOR] RowPtr"); CUDA_CHECK_ERROR()
	//GPUSparse::printGpuArray(d_colIdx, nnz, "[ESTIMATOR] colidx"); CUDA_CHECK_ERROR()

	//x-vector
	float* d_x;
	hipMalloc((void**) &d_x, dim * sizeof(float));

	//b-vector
	float* d_b;
//	hipMalloc((void**) &d_b, dim * sizeof(float));

	//*****************************************************
	// TODO directly obtain device pointers from GPUSparseB
//	d_values = T_sparse->getValueArr(true);

//	int* colIdx = T_sparse->getColIdx();
//	printf("[CMESTIMATOR]: ColIdx\n");
//	Tester::printArrayInt(colIdx, nnz);
//	hipMalloc((void**) &d_colIdx, nnz * sizeof(int));
//	hipMemcpy(d_colIdx, colIdx, nnz * sizeof(int), hipMemcpyHostToDevice);

//	int* rowPtr = T_sparse->getRowPtr();
//	printf("[CMESTIMATOR]: rowPtr\n");
//	Tester::printArrayInt(rowPtr, dim+1);
//	hipMalloc((void**) &d_rowPtr, (dim+1) * sizeof(int));
//	hipMemcpy(d_rowPtr, rowPtr, (dim+1) * sizeof(int), hipMemcpyHostToDevice);
	// END *************************************************

//	printf("pointers\n");

	//Set up cula
	//TODO Try to move to constructor

	culaSparseHandle handle;
	culaSparsePlan plan;
	culaSparseConfig config;

	config.relativeTolerance = 1e-6;
	config.maxIterations = 300;
	config.maxRuntime = 10;

	culaSparseCreate(&handle); //create library handle
	culaSparseConfigInit(handle, &config); //initialize values
	culaSparseCreatePlan(handle, &plan); //create execution plan
	culaSparseSetCudaDevicePlatform(handle, plan, 0); //use the CUDA-device platform (interprets given pointer as device pointers)
	culaSparseSetCgSolver(handle, plan, 0); //associate CG solver with the plan
	culaSparseSetJacobiPreconditioner(handle, plan, 0); //associate jacobi preconditioner with the plan


	int determinedIndicesByNow = 0;
	printf("[CMESTIMATOR]: Solve Eq. system column by column.\n");
	for(int i = 0; i < dim && countIndices < kBest; i++) //if enough values are gathered, stop computation
	{
		//0. determine number of best values for this column
		//The bigger i, the less best indices are determined for this column
		int xBestForThisColumn = ((dim-i)/(0.5*dim*(dim-1))) * kBest;
		if (!xBestForThisColumn) xBestForThisColumn = 1; //at least 1 per column
		//take into account that probably not as many indices as needed can be determined, so try o get them in the next column
		int determineXforThisColumn = xBestForThisColumn + (determinedIndicesByNow - countIndices);
		Indices* tmpIndices = new Indices[determineXforThisColumn];
//		printf("[CMESTIMATOR]: Starting upload\n");
		//1. Compute confidence measure for this column (solve Ax=b)
//		printf("[CMESTIMATOR]: b_column\n");
		d_b = T_sparse->getColumn(i);
//		Tester::printArrayFloat(T_sparse->getColumn(i), dim);
//		hipMemcpy(d_b, T_sparse->getColumn(i), dim * sizeof(float), hipMemcpyHostToDevice);
//		printf("[CMESTIMATOR]: Uploaded column %i.\n",i);

		//debug
//		T_sparse->print();
//		GPUSparse::printGpuArrayF(d_values, nnz, "[ESTIMATOR] Values"); CUDA_CHECK_ERROR()
//		GPUSparse::printGpuArray(d_rowPtr, dim+1, "[ESTIMATOR] RowPtr"); CUDA_CHECK_ERROR()
//		GPUSparse::printGpuArray(d_colIdx, nnz, "[ESTIMATOR] colidx"); CUDA_CHECK_ERROR()
//		GPUSparse::printGpuArrayF(d_b, dim, "[ESTIMATOR] b"); CUDA_CHECK_ERROR()

		computeConfidenceMeasure(handle, plan, config, dim, nnz, d_values, d_rowPtr, d_colIdx, d_x, d_b);
//		printf("[CMESTIMATOR]: Solved column %i.\n",i);
		CUDA_CHECK_ERROR()

		//2. get indices of x best confidence measure values
		if (newMethod)
		{
			int writtenIndices = determineBestConfMeasures(d_x, d_b, i, dim, kBest, determineXforThisColumn, countIndices);
			countIndices += writtenIndices;
		}
		else
		{
			//todo remove me
			tmpIndices = getKBestConfMeasures(d_x, d_b, i, dim, determineXforThisColumn);
			//3. gather indices //todo remove me
			for(int j = 0; j < determineXforThisColumn && countIndices < kBest; j++)
			{
				if (-1 == tmpIndices[j].i) break; //following indices are also -1
				else
				{
					bestIndices[countIndices] = tmpIndices[j];
					countIndices++;
				}
			}
		}

		determinedIndicesByNow += xBestForThisColumn; // #indices that should have been determined

//		printf("Column %i, try to determine %i best values. Actually determined by now %i values\n", i, determineXforThisColumn, countIndices);
	}

	if (newMethod)
	{
		//sort first index array and second index array respectively
		//wrap device pointers
		thrust::device_ptr<int> dp_idx1 = thrust::device_pointer_cast(d_idx1);
		thrust::device_ptr<int> dp_idx2 = thrust::device_pointer_cast(d_idx2);
		CUDA_CHECK_ERROR();

		thrust::sort_by_key(dp_idx1, dp_idx1 + kBest, dp_idx2); //ascending
		CUDA_CHECK_ERROR();

		/*And of course, you can get your raw pointers back if you need to use them in a regular CUDA kernel afterward:
		unsigned int * raw_ptr = thrust::raw_pointer_cast(dev_data_ptr);
		*/
	}

	if (newMethod) //debug printing
	{
		int* h_idx1 = new int[kBest];
		int* h_idx2 = new int[kBest];
		int* h_res = new int[kBest];

		hipMemcpy(h_idx1, d_idx1, kBest*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_idx2, d_idx2, kBest*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(h_res, d_res, kBest*sizeof(int), hipMemcpyDeviceToHost);

		printf("Images to be compared:")
		Tester::printArrayInt(h_idx1, kBest);
		Tester::printArrayInt(h_idx2, kBest);
		Tester::printArrayInt(h_res, kBest);
	}

	//clean up the mess
	hipFree(d_x);
	hipFree(d_b);
	culaSparseDestroyPlan(plan);
	culaSparseDestroy(handle);

	//print
	if (!newMethod)
	{
		printf("%i best entries:\n", kBest);
		for(int i = 0; i < kBest; i++)
		{
			//value can't be printed because it is not saved in the Indices-list
			printf("%i: at [%i,%i]\n",i,bestIndices[i].i,bestIndices[i].j);
		}
	}

	return bestIndices;
}

//handles only device pointer.
void CMEstimatorGPUSparse::computeConfidenceMeasure(culaSparseHandle handle, culaSparsePlan plan, culaSparseConfig config,
															unsigned int dim, unsigned int nnz, float* A, int* rowPtr, int* colIdx, float* x, float* b)
{
	// information returned by the solver
	culaSparseResult result;

	// associate coo data with the plan
	culaSparseSetScsrData(handle, plan, 0, dim, nnz, A, rowPtr, colIdx, x, b);
	CUDA_CHECK_ERROR()

	// execute plan
	culaSparseStatus status = culaSparseExecutePlan(handle, plan, &config, &result);
	CUDA_CHECK_ERROR()

	//print if error
	if (culaSparseNoError != status || false)
	{
		char buffer[512];
		culaSparseGetResultString(handle, &result, buffer, 512);
		CUDA_CHECK_ERROR()
		printf("%s\n", buffer);
	}

	//print resulting vector x if needed
	if (false)
	{
		float* h_x = new float[dim];
		hipMemcpy(h_x, x, dim * sizeof(float), hipMemcpyDeviceToHost);
		CUDA_CHECK_ERROR()
		printf("X = [");
		for (int i = 0; i < dim; i++)
		{
			printf(" %f ", h_x[i]);
		}
		printf("]\n");
	}
}

