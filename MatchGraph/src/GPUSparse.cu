#include "hip/hip_runtime.h"
/*
 * GPUSparse.cpp
 *
 *  Created on: Jun 12, 2013
 *      Author: Armin Gufler
 */

#include "GPUSparse.h"
#include <stdio.h>
#include <stdlib.h>
#include "Tester.h"
#include "Helper.h"
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <fstream>

#define CHECK_FOR_CUDA_ERROR 0

#define CUDA_CHECK_ERROR() {							\
hipError_t err = hipGetLastError();					\
if (hipSuccess != err) {						\
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
            __FILE__, __LINE__, hipGetErrorString(err) );	\
    exit(EXIT_FAILURE);						\
}									\
}

__device__ __constant__ double lambda_times_dim;

/***************************************************************************************/
/**************** KERNELS TO GET THE VALUE ARRAY ***************************************/
__global__ void scatterKernelDouble(double* dst, int num)
{
	//number of this thread
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	for(int pos = i; pos < num; pos += blockDim.x)
	{
		dst[pos] = (-1)*lambda_times_dim;
	}
	//TODO split into a known number of blocks, let each thread write one number

}

__global__ void scatterDiagonalKernelDouble(double* gpuValues, int* gpuRowPtr,
		int* gpuDegrees, int* gpuDiagPos, int dim)
{
	//number of this thread
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < dim)
	{
		int row = i;
		const int valueIndex = gpuRowPtr[row] + gpuDiagPos[row];
		const double valToWrite = 1 + (lambda_times_dim * gpuDegrees[row]);
		gpuValues[valueIndex] = valToWrite;
	}
}
/***************************************************************************************/
/***************************************************************************************/

__global__ void colIdxIncrementKernel(int* colIdx, int* oldColIdx, int* rowPtr, int* incr, int size)
{
	//TODO can this kernel make profitable use of shared memory?

	const unsigned int row = blockIdx.x * gridDim.x + blockIdx.y;

	if (row < size)
	{
		const unsigned int rowIdx = threadIdx.x;

		for (int j = rowIdx; j < (rowPtr[row + 1] - rowPtr[row]); j += blockDim.x)
		{
			colIdx[rowPtr[row] + j + incr[row]] = oldColIdx[rowPtr[row] + j];
		}
		//Note: a single thread has to do more than one step only if blockDim.x < length of row
		// Number of blocks has to be >= number of rows for this kernel!
	}


}

__global__ void arrayAddKernel(int* res, int* _a1, int* _a2, int size)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < size)
	{
		res[i] = _a1[i] + _a2[i];
	}
}

__global__ void rowIncrementedKernel(int* dst, int* idxArray1, int* idxArray2, int dstSize, int idxSize)
{
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

	if(i < idxSize)
	{
		int curIdx = idxArray1[i];
		atomicAdd(&dst[curIdx+1], 1);
		curIdx = idxArray2[i];
		atomicAdd(&dst[dstSize+curIdx+1], 1);
	}
}

__global__ void initKernel(int * array, const int val, const int nwords)
{
    int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

    for(; tIdx < nwords; tIdx += blockDim.x)
        array[tIdx] = val;
}

__global__ void cleanIndexArrays(int* cleanIdx1, int* cleanIdx2, int* negIdx, int* idx1, int* idx2, int* res, int* prefix, int k)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

	if(tIdx < k)
	{
		if(res[tIdx] != 0)
		{
			const int idxToWrite = prefix[tIdx];
			cleanIdx1[idxToWrite] = idx1[tIdx];
			cleanIdx2[idxToWrite] = idx2[tIdx];
		}
		else
		{
			const int idxToWrite = (tIdx - prefix[tIdx])*2;
			negIdx[idxToWrite] = idx1[tIdx];
			negIdx[idxToWrite+1] = idx2[tIdx];
		}
	}
}

__global__ void doInsertionKernel(int* colIdx, int* rowPtr, int* oldRowPtr, int* idxData, int* idxPrefix, int dim)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;
	
	//using one thread per row
	//each thread simply applies insertion sort for each element (sequential)

	if(tIdx < dim)
	{
	  const int start = idxPrefix[tIdx];
	  const int end = idxPrefix[tIdx+1];
	  
	  int counter = 0;
	  for(int i = start; i < end; ++i, counter++)
	  {
		const int elem = idxData[i];
		int j = rowPtr[tIdx] + counter + (oldRowPtr[tIdx+1]-oldRowPtr[tIdx]);
		while(j > rowPtr[tIdx] && colIdx[j-1] > elem)
		{
			colIdx[j] = colIdx[j-1];
			j--;
		}
		  colIdx[j] = elem;
	  }
	}
}

__global__ void initColIdxRowPtrKernel(int* colIdx, int* rowPtr, int dim)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;
	if(tIdx < dim)
	{
		colIdx[tIdx] = tIdx;
		rowPtr[tIdx] = tIdx;
	}
	if(tIdx == 0)
		rowPtr[dim] = dim;
}

__global__ void updateDegreeKernel(int* degrees, int* incr, int dim)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;
	if (tIdx < dim)
	{
		degrees[tIdx] += incr[tIdx+1];
	}
}

__global__ void updateDiagPosKernel(int* diagPos, int* idx1, int* idx2, int size)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;
	if(tIdx < size)
	{
		int col = idx2[tIdx];
		int row = idx1[tIdx];
		if(col < row)
			atomicAdd(&diagPos[row], 1);

		const int tmpSwap = col;
		col = row;
		row = tmpSwap;
		if(col < row)
			atomicAdd(&diagPos[row], 1);
	}
}

__global__ void columnWriteKernelDouble(double* dst, int* colIdx, int* rowPtr, int* numOnes, int col, int dim)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

	if(tIdx == 0){
		//put into 'shared' memory the amount of 1's of this column
		numOnes[0] = rowPtr[col+1] - rowPtr[col];
	}
	__syncthreads();

	for(int i = tIdx; i < numOnes[0] && i < dim; i+=blockDim.x)
	{
		const int idx_in_colIdx = rowPtr[col]+i;
		dst[colIdx[idx_in_colIdx]] = 1.0;
	}

	__syncthreads();
	if(tIdx == 0)
	{
		//diagonal to zero
		dst[col] = 0.0;
	}
}

__global__ void setupCurandkernel(hiprandState *state, int seed)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    /* (from CUDA CURAND documentation) */
    hiprand_init(seed, tid, 0, &state[tid]);
}


//kernel for randomly filling index arrays with not yet compared images
//TODO can for sure be implemented in a much more elegant way ;)
__global__ void randomComparisonFillKernel(int* idx1, int* idx2, int* rowPtr,
							int* colIdx, const int k, const int dim, hiprandState* state)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

	if(tIdx < k)
	{
		int myRandomRow;
		int rangeSize = dim/k;
		const int myLowLimit = tIdx * rangeSize; //rand must be >= this limit
		const int myUpperLimit = myLowLimit + rangeSize - 1; //rand must be < this limit


		hiprandState localState = state[tIdx];
		if(myLowLimit == myUpperLimit)
		{
			myRandomRow = myLowLimit;
		}
		else
		{
			float uniRand = hiprand_uniform(&localState);
			state[tIdx] = localState; //back to global

			myRandomRow = myLowLimit + (uniRand * (myUpperLimit - myLowLimit)) - 1; // should be in [low,up)
		}

		idx1[tIdx] = myRandomRow;

		int currentlyFound = dim+1;
		int last = -1;
		const int minElem = myRandomRow+1; //only lower diagonal matrix allowed
		for(int j = rowPtr[myRandomRow]; j < rowPtr[myRandomRow+1]; j++)
		{
			if(colIdx[j] != last+1)
			{
				//an empty spot found

				//check if not diagonal element (i,i) and >= min
				if(last+1 != myRandomRow && last+1 >= minElem){
					currentlyFound = last+1;
				}

			}
			last = colIdx[j];
		}
		
		//with high probability run on... at larger elments
		if(((hiprand_uniform(&localState) < .80) && (last < dim-1)) || ((last < dim-1) && currentlyFound==dim+1))
		{
		float diff = dim - minElem;
		float breakProb = 1/ diff;
			for(int j = last; j < dim; j++)
			{
				if(j != myRandomRow && j >= minElem)
				{
					currentlyFound = j;
					if(hiprand_uniform(&localState) < breakProb)
					  break;
					breakProb += 1/diff;
				}
			}
		}
		idx2[tIdx] = currentlyFound;
	}
}

//just checks if no values > dim+1 or < 0 are in given array
__global__ void checkRandoms(int* idx1, const int k, const int dim, int* errors)
{
	int tIdx = threadIdx.x + blockDim.x * blockIdx.x;

	if(tIdx < k)
	{
		if(idx1[tIdx] > dim+1 || idx1[tIdx] < 0)
		{
			idx1[tIdx] = dim+1;
			atomicAdd(&errors[0], 1);
		}
	}
}

GPUSparse::GPUSparse(unsigned int _dim, float _lambda) :
		dim(_dim), lambda(_lambda)
{
	num_similar = 0;

	hipMalloc((void**) &_gpuRowPtr, (dim+1)*sizeof(int));
	hipMalloc((void**) &_gpuColIdx, dim*sizeof(int));
	hipMalloc((void**) &_gpuDegrees, dim*sizeof(int));
	hipMalloc((void**) &_gpuDiagPos, dim*sizeof(int));
	hipMalloc((void**) &_gpuColumnPtr, dim*sizeof(double));

	const int numThreads = 128;
	const int numBlocks = 1 + (dim/numThreads);
	initColIdxRowPtrKernel<<<numBlocks, numThreads>>>(_gpuColIdx, _gpuRowPtr, dim);
	initKernel<<<numBlocks, numThreads>>>(_gpuDegrees, 0, dim);
	initKernel<<<numBlocks, numThreads>>>(_gpuDiagPos, 0, dim);
	
	lastRandomK = -1;
}

GPUSparse::~GPUSparse()
{
	hipFree(_gpuColIdx);
	hipFree(_gpuRowPtr);
	hipFree(_gpuDegrees);
	hipFree(_gpuDiagPos);
	hipFree(_gpuColumnPtr);
}

//TODO host thread ?!
void GPUSparse::handleDissimilar(int* idxData, int num)
{
	for(int i = 0; i < num*2; i+=2)
	{
		int col = idxData[i];
		int row = idxData[i+1];

		if(col > dim) //not allowed (safety check)
			break;
		if(col == dim) //avoid diagonal
			continue;

		addDissimilarToColumn(col, row);
		addDissimilarToColumn(row, col);
	}
}

void GPUSparse::updateSparseStatus(int* _idx1, int* _idx2, int* _res, int _k)
{
	const bool verbose = false;

	//this conversion is done just for testing convenience
	int* idx1 = _idx1;
	int* idx2 = _idx2;
	int* res = _res;
	int k = _k;

	//get number of non-zero elements before update
	const int nnzBeforeUpdate = getNNZ();

	if (verbose)
	{
		print();
		printf("New Input for update:\n");
		Helper::printGpuArray(idx1, k, "idx1");
		Helper::printGpuArray(idx2, k, "idx2");
		Helper::printGpuArray(res, k, "res ");
		printf("\n");
	}

	//used for kernel configuration
	int numThreads;
	int numBlocks;

	int* prefixSumResult;
	hipMalloc((void**) &prefixSumResult, (k) * sizeof(int));
	thrust::device_ptr<int> dev_ptr_prefix_res = thrust::device_pointer_cast(prefixSumResult);
	thrust::device_ptr<int> dev_ptr_prefix = thrust::device_pointer_cast(res);
	thrust::exclusive_scan(dev_ptr_prefix, dev_ptr_prefix + k, dev_ptr_prefix_res);
	
#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif	
	
	//TODO move numSimilar computation to GPU, directly in kernel with argument?
	int numSimilar;
	int lastResVal;
	hipMemcpy(&numSimilar, prefixSumResult + (k-1), sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&lastResVal, res + (k-1), sizeof(int), hipMemcpyDeviceToHost);

	if(lastResVal == 1)
		numSimilar++;

//	printf("numSimilar = %i \n", numSimilar);

	int* cleanedIdx;
	int* negativeIndx;
	hipMalloc((void**) &cleanedIdx, numSimilar * 2 * sizeof(int));
	hipMalloc((void**) &negativeIndx, (2*(k - numSimilar))*sizeof(int));
	int* cleanedIdx1 = cleanedIdx;
	int* cleanedIdx2 = cleanedIdx + numSimilar;
	
	numThreads = 32;
	numBlocks = 1 + (k / numThreads);
	cleanIndexArrays<<<numBlocks, numThreads>>>(cleanedIdx1, cleanedIdx2, negativeIndx, idx1, idx2, res, prefixSumResult, k);

	hipFree(prefixSumResult);

	//Handling dissimilar results
	int* negativeIdxHost = (int*) malloc((2*(k - numSimilar))*sizeof(int));
	hipMemcpy(negativeIdxHost, negativeIndx, (2*(k - numSimilar))*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(negativeIndx);

	handleDissimilar(negativeIdxHost, k-numSimilar);
	

	k = numSimilar;
	int* rowData;

	hipMalloc((void**) &rowData, 4 * (dim + 1) * sizeof(int));
	initKernel<<<512, 256>>>(rowData, 0, 4 * (dim + 1));

	int* rowIncrIdx1 = rowData;
	int* rowIncrIdx2 = rowData + (dim + 1);
	int* rowIncr = rowData + 2 * (dim + 1);
	int* prefixIndex1 = rowData + 3 * (dim + 1);

	int* prefixRowIncr; //will be new rowPtr
	hipMalloc((void**) &prefixRowIncr, (dim + 1) * sizeof(int));

	numThreads = 32;
	numBlocks = 1 + (k / numThreads);
	rowIncrementedKernel<<<numBlocks, numThreads>>>(rowData, cleanedIdx1, cleanedIdx2, (dim + 1), k);

	numThreads = 256;
	numBlocks = 1 + ((dim+1)/numThreads);
	arrayAddKernel<<<numBlocks, numThreads>>>(rowIncr, rowIncrIdx1, rowIncrIdx2, (dim + 1));

	dev_ptr_prefix = thrust::device_pointer_cast(rowIncr);
	dev_ptr_prefix_res = thrust::device_pointer_cast(prefixRowIncr);
	thrust::inclusive_scan(dev_ptr_prefix, dev_ptr_prefix + (dim + 1), dev_ptr_prefix_res);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	numThreads = 128;
	numBlocks = 1 + (dim / numThreads);
	updateDegreeKernel<<<numBlocks, numThreads>>>(_gpuDegrees, rowIncr, dim);
	numBlocks = 1 + (k / numThreads);
	updateDiagPosKernel<<<numBlocks, numThreads>>>(_gpuDiagPos, cleanedIdx1, cleanedIdx2, k);

	int* newColIdx;
	const int sizeNewColIdx = (getNNZ() + numSimilar * 2);
	hipMalloc((void**) &newColIdx, sizeNewColIdx * sizeof(int));

	initKernel<<<512, 256>>>(newColIdx, dim + 1, sizeNewColIdx);

	numThreads = 128;
	numBlocks = dim;
	int gridDim = 1 + sqrt(dim);
	dim3 blockGrid(gridDim,gridDim);
	colIdxIncrementKernel<<<blockGrid, numThreads>>>(newColIdx, _gpuColIdx, _gpuRowPtr, prefixRowIncr, dim);


	numThreads = 256;
	numBlocks = 1 + ((dim+1)/numThreads);
	arrayAddKernel<<<numBlocks, numThreads>>>(prefixRowIncr, prefixRowIncr, _gpuRowPtr, (dim + 1));
	

	dev_ptr_prefix = thrust::device_pointer_cast(rowIncrIdx1);
	dev_ptr_prefix_res = thrust::device_pointer_cast(prefixIndex1);
	thrust::inclusive_scan(dev_ptr_prefix, dev_ptr_prefix + (dim + 1), dev_ptr_prefix_res);
	
	numThreads = 32;
	numBlocks = 1 + (dim / numThreads);
	doInsertionKernel<<<numBlocks, numThreads>>>(newColIdx, prefixRowIncr, _gpuRowPtr, cleanedIdx2, prefixIndex1, dim);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//resorting such that sorted after idx2 array
	thrust::device_ptr<int> dpIdx1 = thrust::device_pointer_cast(cleanedIdx1);
	thrust::device_ptr<int> dpIdx2 = thrust::device_pointer_cast(cleanedIdx2);
	thrust::sort_by_key(dpIdx2, dpIdx2 + numSimilar, dpIdx1);

	arrayAddKernel<<<numBlocks, numThreads>>>(_gpuRowPtr, prefixIndex1, _gpuRowPtr, dim + 1);

	dev_ptr_prefix = thrust::device_pointer_cast(rowIncrIdx2);
	thrust::inclusive_scan(dev_ptr_prefix, dev_ptr_prefix + (dim + 1), dev_ptr_prefix_res);

	//insertion sort of new elements on column index
	doInsertionKernel<<<numBlocks, numThreads>>>(newColIdx, prefixRowIncr, _gpuRowPtr, cleanedIdx1, prefixIndex1, dim);

	num_similar += numSimilar;


//	Tester::testCSRMatrixUpdate(Helper::downloadGPUArrayInt(originalRowPtr, dim+1), Helper::downloadGPUArrayInt(_gpuColIdx, oldNNZ), Helper::downloadGPUArrayInt(_gpuDegrees, dim),
//								Helper::downloadGPUArrayInt(prefixRowIncr, dim+1), Helper::downloadGPUArrayInt(newColIdx, getNNZ()),
//								Helper::downloadGPUArrayInt(idx1, _k), Helper::downloadGPUArrayInt(idx2,_k), Helper::downloadGPUArrayInt(res,_k), dissimilarMap, dim, _k);
	//hipFree(originalRowPtr);

	hipFree(_gpuColIdx);
	_gpuColIdx = newColIdx;
	hipFree(_gpuRowPtr);
	_gpuRowPtr = prefixRowIncr;

	hipFree(rowData);
	hipFree(cleanedIdx);
	
	free(negativeIdxHost);

	if (verbose)
	{
		/********* TESTING ****************/
		printf("AFTER UPDATE ---> num similar = %i nnz = %i \n", num_similar, getNNZ());
		Helper::printGpuArray(_gpuColIdx, getNNZ(), "colIdx");
		Helper::printGpuArray(_gpuRowPtr, dim + 1, "rowPtr");
		Helper::printGpuArray(_gpuDiagPos, dim, "diagPos");
		Helper::printGpuArray(_gpuDegrees, dim, "degrees");
		//getValueArr(true);
		//getColumn(3);
		/**********************************/
	}

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif
}

void GPUSparse::set(int i, int j, bool val)
{
	printf("ERROR: set(int, int, bool) not allowed for SPARSE Matrix");
}

void GPUSparse::addDissimilarToColumn(int column, int row)
{
	myElemMap::iterator it = dissimilarMap.find(column);

	if (it == dissimilarMap.end())
	{ //row not in map
		std::set<int> list;
		list.insert(row);
		dissimilarMap.insert(std::pair<int, std::set<int> >(column, list));
	}
	else
	{
		it->second.insert(row);
	}
}

unsigned int GPUSparse::getDimension()
{
	return dim;
}

float* GPUSparse::getConfMatrixF()
{
	return NULL;
}

double* GPUSparse::getValueArrayDouble(bool gpuPointer) const
{
	bool verbose = false;

	int nnz = getNNZ();

	double _cpuLambda_times_dim_double = dim * lambda; //FIXME do not compute each time
	hipMemcpyToSymbol(HIP_SYMBOL(lambda_times_dim), &_cpuLambda_times_dim_double, sizeof(double));

	double* gpuValues;
	hipMalloc((void**) &gpuValues, nnz * sizeof(double));
	int NUM_THREADS = 512;
	int NUM_BLOCKS = 1;
	scatterKernelDouble<<<NUM_BLOCKS, NUM_THREADS>>>(gpuValues, nnz);

	NUM_THREADS = 512;
	NUM_BLOCKS = 1 + (dim / NUM_THREADS);
	scatterDiagonalKernelDouble<<<NUM_BLOCKS, NUM_THREADS>>>(gpuValues, _gpuRowPtr, _gpuDegrees, _gpuDiagPos, dim);

	if (!gpuPointer)
	{
		double* valuesCPU = (double*) malloc(nnz * sizeof(double));
		hipMemcpy(valuesCPU, gpuValues, nnz * sizeof(double), hipMemcpyDeviceToHost);

		hipFree(gpuValues);;

		return valuesCPU;
	}

	if (gpuPointer && verbose)
	{
		double* valuesCPU = (double*) malloc(nnz * sizeof(double));
		hipMemcpy(valuesCPU, gpuValues, nnz * sizeof(double), hipMemcpyDeviceToHost);

		printf("Value array: ");
		Tester::printArray(valuesCPU, nnz);
	}

//	double* valuesCPU = (double*) malloc(nnz * sizeof(double));
//				hipMemcpy(valuesCPU, gpuValues, nnz * sizeof(double), hipMemcpyDeviceToHost);
//		Tester::testValueArray(Helper::downloadGPUArrayInt(_gpuRowPtr, dim+1), Helper::downloadGPUArrayInt(_gpuColIdx, nnz), Helper::downloadGPUArrayInt(_gpuDegrees, dim),
//							dim, nnz, lambda, valuesCPU);

	return gpuValues;
}


double* GPUSparse::getColumnDouble(int columnIdx) const
{
	double* column = new double[dim];
	std::fill_n(column, dim, 0.0f);

	myElemMap::const_iterator it = dissimilarMap.find(columnIdx);

	if (it != dissimilarMap.end())
	{
		std::set<int> dis = it->second;
		for (std::set<int>::const_iterator lIter = dis.begin();
				lIter != dis.end(); ++lIter)
		{
			int idx = (*lIter);
			column[idx] = -1.0f;
		}
	}

	int* _gpuNumOnes;
	hipMalloc((void**) &_gpuNumOnes, sizeof(int));
	hipMemcpy(_gpuColumnPtr, column, dim*sizeof(double), hipMemcpyHostToDevice);

	const int numThreads = 512;
	const int numBlocks = 1;
	columnWriteKernelDouble<<<numBlocks, numThreads>>>(_gpuColumnPtr, _gpuColIdx, _gpuRowPtr, _gpuNumOnes, columnIdx, dim);

	//testing
	if(false)
	{
		hipMemcpy(column, _gpuColumnPtr, dim*sizeof(double), hipMemcpyDeviceToHost);
		Tester::testColumn(dissimilarMap, Helper::downloadGPUArrayInt(_gpuRowPtr, dim+1), Helper::downloadGPUArrayInt(_gpuColIdx, getNNZ()),
					columnIdx, dim, column);
	}

	hipFree(_gpuNumOnes);
	delete[] column;

	return _gpuColumnPtr;
}

char* GPUSparse::getMatrAsArray()
{
	printf("ERROR: getMatrAsArray() not supported by SPARSE Matrix (return NULL)\n");
	return NULL;
}

char GPUSparse::getVal(int i, int j)
{
	printf("ERROR: getVal(int, int) to get a single value not supported by SPARSE Matrix \n");
	return -1;
}

int GPUSparse::getSimilarities()
{
	return getNNZ() - dim;
}

void GPUSparse::print()
{
	printf("###INFO about SPARSE MATRIX###\n");
	printf("dim = %i, lambda = %f, nnz = %i \n", dim, lambda, getNNZ());
//	Helper::printGpuArray(_gpuColIdx, getNNZ(), "colIdx (on GPU)");
//	Helper::printGpuArray(_gpuRowPtr, dim+1, "rowPtr (on GPU)");
}

void GPUSparse::writeGML(char * filename, bool similar, bool dissimilar,
		bool potential)
{
	//TODO
}

int* GPUSparse::getRowPtrDevice() const
{
	return _gpuRowPtr;
}

int* GPUSparse::getColIdxDevice() const
{
	return _gpuColIdx;
}

unsigned int GPUSparse::getNNZ() const
{
	return num_similar * 2 + dim;
}


//TODO not complete yet.
void GPUSparse::fillRandomCompareIndices(int* idx1, int* idx2, int* res, const int k)
{
//	Helper::printGpuArray(_gpuColIdx, getNNZ(), "colIdx (on GPU)");
//	Helper::printGpuArray(_gpuRowPtr, dim+1, "rowPtr (on GPU)");
	
	const int numThreads = 256;
	const int numBlocks = 1 + (numThreads/k);

	if(lastRandomK != k)
	{
	  if(lastRandomK != -1)
	  {
		hipFree(devStates);
	  }

	  const int cudaInitSeed = rand() % 1234567;
	  
	  hipMalloc((void **)&devStates, numThreads * numBlocks *  sizeof(hiprandState));
	  setupCurandkernel<<<numBlocks, numThreads>>>(devStates, cudaInitSeed);
	}
	
	lastRandomK = k;

	initKernel<<<256, 256>>>(res, 0, k); //TODO necessary to initialize res ?

	initKernel<<<256, 256>>>(idx1, dim+1, k);

	initKernel<<<256, 256>>>(idx2, dim+1, k);

	hipDeviceSynchronize();
	
	randomComparisonFillKernel<<<numBlocks, numThreads>>>(idx1, idx2, _gpuRowPtr, _gpuColIdx, k, dim, devStates);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	int* h_idx1 = Helper::downloadGPUArrayInt(idx1, k);
	int* h_idx2 = Helper::downloadGPUArrayInt(idx2, k);
	
	for(int i = 0; i < k; i++)
	{
	  myElemMap::const_iterator it = dissimilarMap.find(h_idx1[i]);
	  if (it != dissimilarMap.end())
	  {
		if(it->second.find(h_idx2[i]) != it->second.end())
		{
		  h_idx1[i] = dim+1;
		  h_idx2[i] = dim+1;
		}
	  }
	}
	
	hipMemcpy(idx1, h_idx1, k*sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(idx2, h_idx2, k*sizeof(int), hipMemcpyHostToDevice);
	
	
	/* ERROR checking
	int* errors;
	hipMalloc((void**) &errors, sizeof(int));
	hipMemset(errors, 0, sizeof(int));
	checkRandoms<<<numBlocks, numThreads>>>(idx1, k, dim, errors);
	int h_errors;
	hipMemcpy(&h_errors, errors, sizeof(int),  hipMemcpyDeviceToHost);
	if(h_errors != 0){
		printf("Error on random fill [idx1]! Aborting...");
		exit(EXIT_FAILURE);
	}
	hipMemset(errors, 0, sizeof(int));
	checkRandoms<<<numBlocks, numThreads>>>(idx1, k, dim, errors);
	hipMemcpy(&h_errors, errors, sizeof(int),  hipMemcpyDeviceToHost);
	if(h_errors != 0){
		printf("Error on random fill [idx2]! Aborting...");
		exit(EXIT_FAILURE);
	}
	*/

	thrust::device_ptr<int> dpIdx1 = thrust::device_pointer_cast(idx1);
	thrust::device_ptr<int> dpIdx2 = thrust::device_pointer_cast(idx2);
	thrust::sort_by_key(dpIdx1, dpIdx1 + k, dpIdx2);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	//free stuff
	free(h_idx1);
	free(h_idx2);

//	Helper::printGpuArray(idx1, k, "i:");
//	Helper::printGpuArray(idx2, k, "j:");
}

void GPUSparse::logSimilarToFile(const char *path, ImageHandler* iHandler) const
{
	std::ofstream file;
	file.open(path);

	file << iHandler->getDirectoryPath() << "\n";

	const char* f1;
	const char* f2;

	for(myElemMap::const_iterator it = dissimilarMap.begin();
			it != dissimilarMap.end(); it++)
	{
		const int column = it->first;
		f1 = iHandler->getImage(column);
		std::set<int> dis = it->second;
		for (std::set<int>::const_iterator lIter = dis.begin(); lIter != dis.end(); ++lIter)
		{
			f2 = iHandler->getImage(*lIter);
			file << f1 << "\t" << f2 << "\t-1\n";
		}
	}

	int* h_rowPtr = Helper::downloadGPUArrayInt(_gpuRowPtr, dim+1);
	int* h_colIdx = Helper::downloadGPUArrayInt(_gpuColIdx, getNNZ());

	for(int i = 0; i < dim; i++)
	{
		const int start = h_rowPtr[i];
		const int end = h_rowPtr[i+1];
		f2 = iHandler->getImage(i);
		for(int idx = start; idx < end; idx++)
		{
			const int column = h_colIdx[idx];
			if(column == i)
				continue;
			f1 = iHandler->getImage(column);
			file << f1 << "\t" << f2 << "\t1\n";
		}
	}

	free(h_rowPtr);
	free(h_colIdx);

	file.close();
}

