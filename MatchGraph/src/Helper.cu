/*
 * Helper.cpp
 *
 * This class contains several static helper functions to be used
 * in this project.
 *
 *  Created on: Jul 10, 2013
 *      Author: Armin, Fabian
 */

#include "Helper.h"
#include "Tester.h"
#include <iostream>
#include <stdio.h>

#define CHECK_FOR_CUDA_ERROR 0

#define CUDA_CHECK_ERROR() {							\
hipError_t err = hipGetLastError();					\
if (hipSuccess != err) {						\
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
            __FILE__, __LINE__, hipGetErrorString(err) );	\
    exit(EXIT_FAILURE);						\
}									\
}

/*
 * Upload data from host to device.
 */
void Helper::cudaMemcpyArrayInt(int* h_src, int* d_trg, int size)
{
	hipMemcpy(d_trg, h_src, size*sizeof(int), hipMemcpyHostToDevice);
}

/*
 * Download data from device to host.
 */
void Helper::cudaMemcpyArrayIntToHost(int* d_src, int* h_trg, int size)
{
	hipMemcpy(h_trg, d_src, sizeof(int)*size, hipMemcpyDeviceToHost);
}

/*
 * Print an integer device array on console with a given message.
 */
void Helper::printGpuArray(int * devPtr, const int size, std::string message)
{
	int* cpu = (int*) malloc(sizeof(int)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(int), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayInt(cpu, size);
	free(cpu);
}

/*
 * Print a float device array on console with a given message.
 */
void Helper::printGpuArrayF(float * devPtr, const int size, std::string message)
{
	float* cpu = (float*) malloc(sizeof(float)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(float), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayFloat(cpu, size);
	free(cpu);
}

/*
 * Print a double device array on console with a given message.
 */
void Helper::printGpuArrayD(double * devPtr, const int size, std::string message)
{
	double* cpu = (double*) malloc(sizeof(double)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(double), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayDouble(cpu, size);
	free(cpu);
}

/*
 * Print a long device array on console with a given message.
 */
void Helper::printGpuArrayL(long * devPtr, const int size, std::string message)
{
	long* cpu = (long*) malloc(sizeof(long)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(long), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayLong(cpu, size);
	free(cpu);
}

/*
 * Download given device memory location to host and return a pointer to it.
 */
int* Helper::downloadGPUArrayInt(int* devPtr, const int size)
{
	int* cpu = (int*) malloc(sizeof(int)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(int), hipMemcpyDeviceToHost);
	return cpu;
}

/*
 * Download given device memory location to host and return a pointer to it.
 */
float* Helper::downloadGPUArrayFloat(float* devPtr, const int size)
{
	float* cpu = (float*) malloc(sizeof(float)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(float), hipMemcpyDeviceToHost);
	return cpu;
}

/*
 * Download given device memory location to host and return a pointer to it.
 */
double* Helper::downloadGPUArrayDouble(double* devPtr, const int size)
{
	double* cpu = (double*) malloc(sizeof(double)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(double), hipMemcpyDeviceToHost);
	return cpu;
}
