/*
 * Helper.cpp
 *
 *  Created on: Jul 10, 2013
 *      Author: schwarzk
 */

#include "Helper.h"
#include "Tester.h"
#include <iostream>
#include <stdio.h>

#define CHECK_FOR_CUDA_ERROR 1

#define CUDA_CHECK_ERROR() {							\
hipError_t err = hipGetLastError();					\
if (hipSuccess != err) {						\
    fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
            __FILE__, __LINE__, hipGetErrorString(err) );	\
    exit(EXIT_FAILURE);						\
}									\
}

void Helper::cudaMemcpyArrayInt(int* h_src, int* d_trg, int size)
{
	hipMemcpy(d_trg, h_src, size*sizeof(int), hipMemcpyHostToDevice);
}

void Helper::printGpuArray(int * devPtr, const int size, std::string message)
{
	int* cpu = (int*) malloc(sizeof(int)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(int), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayInt(cpu, size);
	free(cpu);
}

void Helper::printGpuArrayF(float * devPtr, const int size, std::string message)
{
	float* cpu = (float*) malloc(sizeof(float)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(float), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayFloat(cpu, size);
	free(cpu);
}

void Helper::printGpuArrayD(double * devPtr, const int size, std::string message)
{
	double* cpu = (double*) malloc(sizeof(double)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(double), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayDouble(cpu, size);
	free(cpu);
}

void Helper::printGpuArrayL(long * devPtr, const int size, std::string message)
{
	long* cpu = (long*) malloc(sizeof(long)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(long), hipMemcpyDeviceToHost);

#if CHECK_FOR_CUDA_ERROR
	CUDA_CHECK_ERROR()
#endif

	std::cout << message << " : ";
	Tester::printArrayLong(cpu, size);
	free(cpu);
}

int* Helper::downloadGPUArrayInt(int* devPtr, const int size)
{
	int* cpu = (int*) malloc(sizeof(int)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(int), hipMemcpyDeviceToHost);
	return cpu;
}

float* Helper::downloadGPUArrayFloat(float* devPtr, const int size)
{
	float* cpu = (float*) malloc(sizeof(float)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(float), hipMemcpyDeviceToHost);
	return cpu;
}

double* Helper::downloadGPUArrayDouble(double* devPtr, const int size)
{
	double* cpu = (double*) malloc(sizeof(double)*size);
	hipMemcpy(cpu, devPtr, size*sizeof(double), hipMemcpyDeviceToHost);
	return cpu;
}
