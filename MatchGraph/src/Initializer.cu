#include "hip/hip_runtime.h"
/*
 * Initializer.cu
 *
 * Initializes a given directory by indexing all images within this directory
 * in a map such that the image-path can be obtained fast by the corresponding
 * index.
 * Initializes the T Matrix.
 *
 *  Created on: Jun 29, 2013
 *      Author: schwarzk
 */

#include "Initializer.h"
#include <iostream>
#include <stdio.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include "Tester.h"
#include "GPUSparse.h"

#define CUDA_CHECK_ERROR() {							\
    hipError_t err = hipGetLastError();					\
    if (hipSuccess != err) {						\
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.",	\
                __FILE__, __LINE__, hipGetErrorString(err) );	\
        exit(EXIT_FAILURE);						\
    }									\
}

const int THREADS = 128;

//Initialize index arrays
static __global__ void initIndexArrays(int* d_idx1, int* d_idx2, int* d_res, int dim, int initArraySize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < dim)
	{
		d_idx1[idx] = idx;
		d_idx2[idx] = idx;
		if (idx < initArraySize) d_res[idx] = 0;
	}
}

//Simple kernel to shuffle an array on GPU (prevents memcpy). To be called only with ONE thread!
static __global__ void shuffleArrayKernel(int* array, int dim, int initArraySize, hiprandState* globalStates, unsigned long seed)
{
	int idx = threadIdx.x;
	hiprand_init(seed, idx, 0, &globalStates[idx]);
	hiprandState localState = globalStates[idx];
	for(int i = 0; i < initArraySize; i++) //just for the needed index array size
	{
		int shift = hiprand_uniform(&localState) * dim;
		int shift_idx = (i+shift)%dim;
		//shuffle
		int tmp = array[i];
		array[i] = array[shift_idx];
		array[shift_idx] = tmp;
	}
}

//Kernel to swap upper diagonal matrix elements
static __global__ void swapUpperDiagonal(int* idx1, int* idx2, int initArraySize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < initArraySize)
	{
		//swap upper diagonal elements
		if (idx1[idx] <= idx2[idx])
		{
			int tmp = idx1[idx];
			idx1[idx] = idx2[idx];
			idx2[idx] = tmp;
		}
	}
}

//check for symmetric entries (duplicates caused by swapping earlier)
static __global__ void checkForDuplicates(int* idx1, int* idx2, int initArraySize, int dim)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < (initArraySize-1))
	{
		if (idx1[idx] == idx2[idx]) //diagonal element (can not occur twice)
		{
			//mask out
			idx1[idx] = dim+1;
			idx2[idx] = dim+1;
		}
		else
		{
			int compareIdx = idx2[idx];
			for(int j = idx+1; j < initArraySize && idx1[idx] == idx1[j]; j++)
			{
				if (compareIdx == idx2[j])
				{
					//mask out
					idx1[idx] = dim+1;
					idx2[idx] = dim+1;
					break; //only one duplicate possible
				}
			}
		}
	}
}


Initializer::Initializer()
{

}

Initializer::~Initializer()
{

}

/*
 * Initializes the T-Matrix with random image comparisons.
 */
void Initializer::doInitializationPhase(MatrixHandler* T, ImageHandler* iHandler, ImageComparator* comparator, int initArraySize)
{
	bool debugPrint = false;

	unsigned int dim = T->getDimension();

	if (initArraySize > dim)
	{
		printf("[INITIALIZER]: initialization array size must be smaller than dimension!\n");
		exit(EXIT_FAILURE);
	}

	//Initialization arrays
	int* d_initIdx1;
	int* d_initIdx2;
	int* d_initRes;
	hipMalloc(&d_initIdx1, dim*sizeof(int));
	hipMalloc(&d_initIdx2, dim*sizeof(int));
	hipMalloc(&d_initRes, initArraySize*sizeof(int));

	//initialize these arrays
	int numBlocks = (dim + THREADS - 1) / THREADS;
	initIndexArrays<<<numBlocks, THREADS>>>(d_initIdx1, d_initIdx2, d_initRes, dim, initArraySize);
	CUDA_CHECK_ERROR()

	int* testResult1 = new int[dim];
	int* testResult2 = new int[dim];
	int* testResult3 = new int[initArraySize];
	//todo remove debug printing
	if (debugPrint)
	{
		printf("Init-Index-Array initialization\n");
		hipMemcpy(testResult1, d_initIdx1, dim*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, dim*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult3, d_initRes, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, dim);
		Tester::printArrayInt(testResult2, dim);
		Tester::printArrayInt(testResult3, initArraySize);
	}

	//initialize cuRAND
	hiprandState* states;
	hipMalloc(&states, sizeof(hiprandState));

	//shuffle Arrays
	shuffleArrayKernel<<<1, 1>>>(d_initIdx1, dim, initArraySize, states, time(NULL));
	CUDA_CHECK_ERROR()
	shuffleArrayKernel<<<1, 1>>>(d_initIdx2, dim, initArraySize, states, 3*time(NULL)); //ensure different seeds
	CUDA_CHECK_ERROR()

	//todo remove debug printing
	if (debugPrint)
	{
		printf("shuffled\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
	}

	//mask each entry on the upper diagonal matrix (prevents symmetrical image comparisons)
	//swap entries inside upper diagonal-matrix (map to lower diagonal matrix)
	numBlocks = (initArraySize + THREADS - 1) / THREADS;
	swapUpperDiagonal<<<numBlocks, THREADS>>>(d_initIdx1, d_initIdx2, initArraySize);
	CUDA_CHECK_ERROR()

	//todo remove debug printing
	if (debugPrint)
	{
		printf("swapped\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
	}

	//sort index array1 ascending and index array2 respectively
	//wrap device pointers
	thrust::device_ptr<int> dp_initIdx1 = thrust::device_pointer_cast(d_initIdx1);
	thrust::device_ptr<int> dp_initIdx2 = thrust::device_pointer_cast(d_initIdx2);
	CUDA_CHECK_ERROR();
	thrust::sort_by_key(dp_initIdx1, dp_initIdx1 + initArraySize, dp_initIdx2); //ascending
	CUDA_CHECK_ERROR();

	//todo remove debug printing
	if (debugPrint)
	{
		printf("sorted\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
	}

	//check for duplicated entries and mask them out
	checkForDuplicates<<<numBlocks, THREADS>>>(d_initIdx1, d_initIdx2, initArraySize, dim);
	CUDA_CHECK_ERROR();

	//todo remove debug printing
	if (debugPrint)
	{
		printf("masked duplicates\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
	}

	//sort again
	thrust::sort_by_key(dp_initIdx1, dp_initIdx1 + initArraySize, dp_initIdx2); //ascending
	CUDA_CHECK_ERROR();

	//todo remove debug printing
	if (debugPrint)
	{
		printf("sorted\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
	}

	//compare images
	comparator->doComparison(iHandler, T, d_initIdx1, d_initIdx2, d_initRes, initArraySize);

	//todo remove debug printing
	if (debugPrint)
	{
		printf("result after comparison -> this will be given to GPUSparse to update the Marix\n");
		hipMemcpy(testResult1, d_initIdx1, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult2, d_initIdx2, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(testResult3, d_initRes, initArraySize*sizeof(int), hipMemcpyDeviceToHost);
		Tester::printArrayInt(testResult1, initArraySize);
		Tester::printArrayInt(testResult2, initArraySize);
		Tester::printArrayInt(testResult3, initArraySize);
	}


	//initialize T Matrix with compared images
	//invoked only on sparse matrixhandler
	GPUSparse* T_sparse = dynamic_cast<GPUSparse*> (T);
	T_sparse->updateSparseStatus(d_initIdx1, d_initIdx2, d_initRes, initArraySize);

	//cleanup after completed initialization
	hipFree(d_initIdx1);
	hipFree(d_initIdx2);
	hipFree(d_initRes);

	delete[] testResult1;
	delete[] testResult2;
	delete[] testResult3;
}


